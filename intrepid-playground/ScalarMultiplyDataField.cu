// -*- C++ -*-
// matrixMultiplication.cc
// a huge comparison of doing naive and tiled matrix multiplication using many
//  different methods and technologies

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <string>
#include <algorithm>

// yucky, but for asking the system how many cores we have
#include <unistd.h>
#include <assert.h>

// header file for openmp
#include <omp.h>

// header files for kokkos
#include <Kokkos_Core.hpp>
#include "Teuchos_Array.hpp"
#include "Intrepid_ArrayTools.hpp"
#include "Intrepid_FieldContainer.hpp"
#include "Intrepid_RealSpaceTools.hpp"
#include "Teuchos_oblackholestream.hpp"
#include "Teuchos_RCP.hpp"
#include "Teuchos_ScalarTraits.hpp"
#include "Teuchos_GlobalMPISession.hpp"

#include <hip/hip_runtime.h>

using std::string;
using std::vector;
using Intrepid::FieldContainer;

typedef Intrepid::RealSpaceTools<double> rst;

#define BLOCK_SIZE 64;

//Pre-C++11 timing (thanks jeff)
double getElapsedTime(const timespec start, const timespec end) {
  timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return double(temp.tv_sec) + double(temp.tv_nsec) / 1e9;
}

template<class ArrayOutFields, class ArrayInData, class ArrayInFields>
void scalarMultiplyDataField(ArrayOutFields &     outputFields,
                                         const ArrayInData &  inputData,
                                         ArrayInFields &      inputFields,
                                         const bool           reciprocal) {

#ifdef HAVE_INTREPID_DEBUG
  TEUCHOS_TEST_FOR_EXCEPTION( (inputData.rank() != 2), std::invalid_argument,
                      ">>> ERROR (ArrayTools::scalarMultiplyDataField): Input data container must have rank 2.");
  if (outputFields.rank() <= inputFields.rank()) {
    TEUCHOS_TEST_FOR_EXCEPTION( ( (inputFields.rank() < 3) || (inputFields.rank() > 5) ), std::invalid_argument,
                        ">>> ERROR (ArrayTools::scalarMultiplyDataField): Input fields container must have rank 3, 4, or 5.");
    TEUCHOS_TEST_FOR_EXCEPTION( (outputFields.rank() != inputFields.rank()), std::invalid_argument,
                        ">>> ERROR (ArrayTools::scalarMultiplyDataField): Input and output fields containers must have the same rank.");
    TEUCHOS_TEST_FOR_EXCEPTION( (inputFields.dimension(0) != inputData.dimension(0) ), std::invalid_argument,
                        ">>> ERROR (ArrayTools::scalarMultiplyDataField): Zeroth dimensions (number of integration domains) of the fields and data input containers must agree!");
    TEUCHOS_TEST_FOR_EXCEPTION( ( (inputFields.dimension(2) != inputData.dimension(1)) && (inputData.dimension(1) != 1) ), std::invalid_argument,
                        ">>> ERROR (ArrayTools::scalarMultiplyDataField): Second dimension of the fields input container and first dimension of data input container (number of integration points) must agree or first data dimension must be 1!");
    for (int i=0; i<inputFields.rank(); i++) {
      std::string errmsg  = ">>> ERROR (ArrayTools::scalarMultiplyDataField): Dimension ";
      errmsg += (char)(48+i);
      errmsg += " of the input and output fields containers must agree!";
      TEUCHOS_TEST_FOR_EXCEPTION( (inputFields.dimension(i) != outputFields.dimension(i)), std::invalid_argument, errmsg );
    }
  }
  else {
    TEUCHOS_TEST_FOR_EXCEPTION( ( (inputFields.rank() < 2) || (inputFields.rank() > 4) ), std::invalid_argument,
                        ">>> ERROR (ArrayTools::scalarMultiplyDataField): Input fields container must have rank 2, 3, or 4.");
    TEUCHOS_TEST_FOR_EXCEPTION( (outputFields.rank() != inputFields.rank()+1), std::invalid_argument,
                        ">>> ERROR (ArrayTools::scalarMultiplyDataField): The rank of the input fields container must be one less than the rank of the output fields container.");
    TEUCHOS_TEST_FOR_EXCEPTION( ( (inputFields.dimension(1) != inputData.dimension(1)) && (inputData.dimension(1) != 1) ), std::invalid_argument,
                        ">>> ERROR (ArrayTools::scalarMultiplyDataField): First dimensions of fields input container and data input container (number of integration points) must agree or first data dimension must be 1!");
    TEUCHOS_TEST_FOR_EXCEPTION( ( inputData.dimension(0) != outputFields.dimension(0) ), std::invalid_argument,
                        ">>> ERROR (ArrayTools::scalarMultiplyDataField): Zeroth dimensions of fields output container and data input containers (number of integration domains) must agree!");
    for (int i=0; i<inputFields.rank(); i++) {
      std::string errmsg  = ">>> ERROR (ArrayTools::scalarMultiplyDataField): Dimensions ";
      errmsg += (char)(48+i);
      errmsg += " and ";
      errmsg += (char)(48+i+1);
      errmsg += " of the input and output fields containers must agree!";
      TEUCHOS_TEST_FOR_EXCEPTION( (inputFields.dimension(i) != outputFields.dimension(i+1)), std::invalid_argument, errmsg );
    }
  }
#endif

  // get sizes
  int invalRank      = inputFields.rank();
  int outvalRank     = outputFields.rank();
  int numCells       = outputFields.dimension(0);
  int numFields      = outputFields.dimension(1);
  int numPoints      = outputFields.dimension(2);
  int numDataPoints  = inputData.dimension(1);
  int dim1Tens       = 0;
  int dim2Tens       = 0;
  if (outvalRank > 3) {
    dim1Tens = outputFields.dimension(3);
    if (outvalRank > 4) {
      dim2Tens = outputFields.dimension(4);
    }
  }

  if (outvalRank == invalRank) {

    if (numDataPoints != 1) { // nonconstant data

      switch(invalRank) {
        case 3: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  outputFields(cl, bf, pt) = inputFields(cl, bf, pt)/inputData(cl, pt);
                } // P-loop
              } // F-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  outputFields(cl, bf, pt) = inputFields(cl, bf, pt)*inputData(cl, pt);
                } // P-loop
              } // F-loop
            } // C-loop
          }
        }// case 3
        break;

        case 4: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iVec = 0; iVec < dim1Tens; iVec++) {
                    outputFields(cl, bf, pt, iVec) = inputFields(cl, bf, pt, iVec)/inputData(cl, pt);
                  } // D1-loop
                } // P-loop
              } // F-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iVec = 0; iVec < dim1Tens; iVec++) {
                    outputFields(cl, bf, pt, iVec) = inputFields(cl, bf, pt, iVec)*inputData(cl, pt);
                  } // D1-loop
                } // P-loop
              } // F-loop
            } // C-loop
          }
        }// case 4
        break;

        case 5: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iTens1 = 0; iTens1 < dim1Tens; iTens1++) {
                    for( int iTens2 = 0; iTens2 < dim2Tens; iTens2++) {
                      outputFields(cl, bf, pt, iTens1, iTens2) = inputFields(cl, bf, pt, iTens1, iTens2)/inputData(cl, pt);
                    } // D2-loop
                  } // D1-loop
                } // F-loop
              } // P-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iTens1 = 0; iTens1 < dim1Tens; iTens1++) {
                    for( int iTens2 = 0; iTens2 < dim2Tens; iTens2++) {
                      outputFields(cl, bf, pt, iTens1, iTens2) = inputFields(cl, bf, pt, iTens1, iTens2)*inputData(cl, pt);
                    } // D2-loop
                  } // D1-loop
                } // F-loop
              } // P-loop
            } // C-loop
          }
        }// case 5
        break;

        default:
	    ;
	}// invalRank

    }
    else { //constant data

      switch(invalRank) {
        case 3: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  outputFields(cl, bf, pt) = inputFields(cl, bf, pt)/inputData(cl, 0);
                } // P-loop
              } // F-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  outputFields(cl, bf, pt) = inputFields(cl, bf, pt)*inputData(cl, 0);
                } // P-loop
              } // F-loop
            } // C-loop
          }
        }// case 3
        break;

        case 4: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iVec = 0; iVec < dim1Tens; iVec++) {
                    outputFields(cl, bf, pt, iVec) = inputFields(cl, bf, pt, iVec)/inputData(cl, 0);
                  } // D1-loop
                } // P-loop
              } // F-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iVec = 0; iVec < dim1Tens; iVec++) {
                    outputFields(cl, bf, pt, iVec) = inputFields(cl, bf, pt, iVec)*inputData(cl, 0);
                  } // D1-loop
                } // P-loop
              } // F-loop
            } // C-loop
          }
        }// case 4
        break;

        case 5: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iTens1 = 0; iTens1 < dim1Tens; iTens1++) {
                    for( int iTens2 = 0; iTens2 < dim2Tens; iTens2++) {
                      outputFields(cl, bf, pt, iTens1, iTens2) = inputFields(cl, bf, pt, iTens1, iTens2)/inputData(cl, 0);
                    } // D2-loop
                  } // D1-loop
                } // F-loop
              } // P-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iTens1 = 0; iTens1 < dim1Tens; iTens1++) {
                    for( int iTens2 = 0; iTens2 < dim2Tens; iTens2++) {
                      outputFields(cl, bf, pt, iTens1, iTens2) = inputFields(cl, bf, pt, iTens1, iTens2)*inputData(cl, 0);
                    } // D2-loop
                  } // D1-loop
                } // F-loop
              } // P-loop
            } // C-loop
          }
        }// case 5
        break;

        default:
	    ; 
      } // invalRank
    } // numDataPoints

  }
  else {

    if (numDataPoints != 1) { // nonconstant data

      switch(invalRank) {
        case 2: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  outputFields(cl, bf, pt) = inputFields(bf, pt)/inputData(cl, pt);
                } // P-loop
              } // F-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  outputFields(cl, bf, pt) = inputFields(bf, pt)*inputData(cl, pt);
                } // P-loop
              } // F-loop
            } // C-loop
          }
        }// case 2
        break;

        case 3: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iVec = 0; iVec < dim1Tens; iVec++) {
                    outputFields(cl, bf, pt, iVec) = inputFields(bf, pt, iVec)/inputData(cl, pt);
                  } // D1-loop
                } // P-loop
              } // F-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iVec = 0; iVec < dim1Tens; iVec++) {
                    outputFields(cl, bf, pt, iVec) = inputFields(bf, pt, iVec)*inputData(cl, pt);
                  } // D1-loop
                } // P-loop
              } // F-loop
            } // C-loop
          }
        }// case 3
        break;

        case 4: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iTens1 = 0; iTens1 < dim1Tens; iTens1++) {
                    for( int iTens2 = 0; iTens2 < dim2Tens; iTens2++) {
                      outputFields(cl, bf, pt, iTens1, iTens2) = inputFields(bf, pt, iTens1, iTens2)/inputData(cl, pt);
                    } // D2-loop
                  } // D1-loop
                } // F-loop
              } // P-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iTens1 = 0; iTens1 < dim1Tens; iTens1++) {
                    for( int iTens2 = 0; iTens2 < dim2Tens; iTens2++) {
                      outputFields(cl, bf, pt, iTens1, iTens2) = inputFields(bf, pt, iTens1, iTens2)*inputData(cl, pt);
                    } // D2-loop
                  } // D1-loop
                } // F-loop
              } // P-loop
            } // C-loop
          }
        }// case 4
        break;

        default:
	    ;
	}// invalRank

    }
    else { //constant data

      switch(invalRank) {
        case 2: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  outputFields(cl, bf, pt) = inputFields(bf, pt)/inputData(cl, 0);
                } // P-loop
              } // F-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  outputFields(cl, bf, pt) = inputFields(bf, pt)*inputData(cl, 0);
                } // P-loop
              } // F-loop
            } // C-loop
          }
        }// case 2
        break;

        case 3: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iVec = 0; iVec < dim1Tens; iVec++) {
                    outputFields(cl, bf, pt, iVec) = inputFields(bf, pt, iVec)/inputData(cl, 0);
                  } // D1-loop
                } // P-loop
              } // F-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iVec = 0; iVec < dim1Tens; iVec++) {
                    outputFields(cl, bf, pt, iVec) = inputFields(bf, pt, iVec)*inputData(cl, 0);
                  } // D1-loop
                } // P-loop
              } // F-loop
            } // C-loop
          }
        }// case 3
        break;

        case 4: {
          if (reciprocal) {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iTens1 = 0; iTens1 < dim1Tens; iTens1++) {
                    for( int iTens2 = 0; iTens2 < dim2Tens; iTens2++) {
                      outputFields(cl, bf, pt, iTens1, iTens2) = inputFields(bf, pt, iTens1, iTens2)/inputData(cl, 0);
                    } // D2-loop
                  } // D1-loop
                } // F-loop
              } // P-loop
            } // C-loop
          }
          else {
            for(int cl = 0; cl < numCells; cl++) {
              for(int bf = 0; bf < numFields; bf++) {
                for(int pt = 0; pt < numPoints; pt++) {
                  for( int iTens1 = 0; iTens1 < dim1Tens; iTens1++) {
                    for( int iTens2 = 0; iTens2 < dim2Tens; iTens2++) {
                      outputFields(cl, bf, pt, iTens1, iTens2) = inputFields(bf, pt, iTens1, iTens2)*inputData(cl, 0);
                    } // D2-loop
                  } // D1-loop
                } // F-loop
              } // P-loop
            } // C-loop
          }
        }// case 4
        break;

        default:
	    ;
	} // invalRank
    } // numDataPoints

  } // end if (outvalRank = invalRank)

}


int main(int argc, char* argv[]) {
    int c = 10000, p = 1000; //l = 10, r = 10, t1 = 10, t2 = 10;
    int b = 100;

    FieldContainer<double> in_Fields_3(c, b, p);
    FieldContainer<double> in_Data_2(c, p);

    FieldContainer<double> out_Fields3_Serial(c, b, p);
    FieldContainer<double> out_Fields3(c, b, p);

    for (int i = 0; i < in_Fields_3.size(); i++) {
	in_Fields_3[i] = Teuchos::ScalarTraits<double>::random();
    }
    for (int i = 0; i < in_Data_2.size(); i++) {
	in_Data_2[i] = Teuchos::ScalarTraits<double>::random();
    }
    std::cout << "Created the vectors" << std::endl;

    std::cout << "Trying serial" << std::endl;

    timespec tic;
    clock_gettime(CLOCK_MONOTONIC, &tic);

    scalarMultiplyDataField<FieldContainer<double>, FieldContainer<double>,
    FieldContainer<double> >(out_Fields3_Serial, in_Data_2, in_Fields_3, false);

    timespec toc;
    clock_gettime(CLOCK_MONOTONIC, &toc);
    const double elapsedTime_serial = getElapsedTime(tic, toc);

    std::cout << "serial took " << elapsedTime_serial << " seconds" <<
    std::endl;
    

    Kokkos::initialize();

    typedef Kokkos::View<double ***, Kokkos::LayoutRight, Kokkos::Cuda>
    cuda_input_fields_3;
    typedef Kokkos::View<double **, Kokkos::LayoutRight, Kokkos::Cuda>
    cuda_input_data_2;

    typedef Kokkos::View<double ***, Kokkos::LayoutRight, Kokkos::Cuda>
    cuda_output_fields_3;

    typedef typename cuda_input_fields_3::HostMirror cuda_input_fields_3_host;
    typedef typename cuda_input_data_2::HostMirror cuda_input_data_2_host;
    typedef typename cuda_output_fields_3::HostMirror cuda_output_fields_3_host;

    /*
    typedef Kokkos::View<double ***, Kokkos::LayoutRight, Kokkos::OpenMP>
    omp_input_view_t;
    typedef Kokkos::View<double ***, Kokkos::LayoutRight, Kokkos::OpenMP>
    omp_output_view_t;
    */

    cuda_input_fields_3 cuda_kokkosInputFields("input_Fields", c, b, p);
    cuda_input_data_2 cuda_kokkosInputData("input_Data", c, p);
    cuda_output_fields_3 cuda_kokkosOut("output", c, b, p);

    cuda_input_fields_3_host cuda_hostFields("left_input", c, b, p);
    cuda_input_data_2_host cuda_hostData("left_input", c, p);
    cuda_output_fields_3_host cuda_hostOut("left_input", c, b, p);

    printf("filling views\n");
    
    for (int cl = 0; cl < c; cl++) {
	for (int pt = 0; pt < p; pt++) {
	    for (int bf = 0; bf < b; bf++) {
		cuda_hostFields(cl, bf, pt) = in_Fields_3(cl, bf, pt);
	    }
	    cuda_hostData(cl, pt) = in_Data_2(cl, pt);
	}
    }
		
    //Now I need to call the function that will create the functor and run!


    Kokkos::finalize();

    return 0;

}








