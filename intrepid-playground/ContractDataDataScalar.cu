#include "hip/hip_runtime.h"
// -*- C++ -*-
// matrixMultiplication.cc
// a huge comparison of doing naive and tiled matrix multiplication using many
//  different methods and technologies

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <string>
#include <algorithm>

// yucky, but for asking the system how many cores we have
#include <unistd.h>

// header file for openmp
#include <omp.h>

// header files for kokkos
#include <Kokkos_Core.hpp>
#include "Teuchos_Array.hpp"
#include "Intrepid_ArrayTools.hpp"
#include "Intrepid_FieldContainer.hpp"
#include "Intrepid_RealSpaceTools.hpp"
#include "Teuchos_oblackholestream.hpp"
#include "Teuchos_RCP.hpp"
#include "Teuchos_ScalarTraits.hpp"
#include "Teuchos_GlobalMPISession.hpp"

#include <hip/hip_runtime.h>

using std::string;
using std::vector;
using Intrepid::FieldContainer;

typedef Intrepid::RealSpaceTools<double> rst;

#define BLOCK_SIZE 64;

//Pre-C++11 timing (thanks jeff)
double getElapsedTime(const timespec start, const timespec end) {
	timespec temp;
	if ((end.tv_nsec-start.tv_nsec)<0) {
		temp.tv_sec = end.tv_sec-start.tv_sec-1;
		temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
	} else {
		temp.tv_sec = end.tv_sec-start.tv_sec;
		temp.tv_nsec = end.tv_nsec-start.tv_nsec;
	}
	return double(temp.tv_sec) + double(temp.tv_nsec) / 1e9;
}


__global__
void
cudaDocontractDataDataScalar_kernelColMajor(double * d_left, double * d_right,
		double * d_out,
		int numCells,
		int numPoints) {

	int myID = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(myID < numCells) {
		double temp = 0;
		for (int qp = 0; qp < numPoints; qp++) {
			temp += d_left[myID + qp*numCells] * d_right[myID + qp*numCells];
		}
		d_out[myID]=temp;
	}
}

__global__
void
cudaDocontractDataDataScalar_kernelRowMajor(double * d_left, double * d_right,
double * d_out,
int numCells,
int numPoints) {

	int myID = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(myID < numCells) {
		double temp = 0;
		for (int qp = 0; qp < numPoints; qp++) {
			temp += d_left[myID*numPoints + qp] * d_right[myID*numPoints + qp];
		}
		d_out[myID]= temp;
	}
}

void
cudaDoContractDataDataScalar(double * h_out,
		double * h_inLeft,
		double * h_inRight,
		int numCells,
		int numPoints,
		bool colMajor) {

	double * d_right;
	double * d_left;
	double * d_out;

	hipMalloc(&d_right, sizeof(double) * numCells  * numPoints);

	hipMalloc(&d_left, sizeof(double) * numCells * numPoints);

	hipMalloc(&d_out, sizeof(double) * numCells);

	hipMemset(d_out, 0, sizeof(double) * numCells);

	hipMemcpy(d_right, h_inRight,
			sizeof(double) * numCells * numPoints, hipMemcpyHostToDevice);

	hipMemcpy(d_left, h_inLeft,
			sizeof(double) * numCells * numPoints, hipMemcpyHostToDevice);


	dim3 blockSize(1024);
	dim3 gridSize((numCells / 1024) + 1);

	if(colMajor)
		cudaDocontractDataDataScalar_kernelColMajor<<<gridSize, blockSize>>>(d_left,
			d_right, d_out, numCells,numPoints);
	else
		cudaDocontractDataDataScalar_kernelRowMajor<<<gridSize, blockSize>>>(d_left,
		d_right, d_out, numCells,numPoints);

	hipMemcpy(h_out, d_out, sizeof(double) * numCells, hipMemcpyDeviceToHost);

  hipFree(d_right);
  hipFree(d_left);
  hipFree(d_out);

}

__global__
void
cudaDocontractDataDataScalar_kernelUnrolled(double * d_left, double * d_right,
double * d_out,
int numCells,
int numPoints) {

	int myID = (blockIdx.x * blockDim.x) + threadIdx.x + (blockIdx.y * numPoints);

	if(myID < numCells * numPoints) {
		temp += d_left[myID*numPoints + qp] * d_right[myID*numPoints + qp];

		d_out[myID]= temp;
	}
}

void
cudaDoContractDataDataScalarUnrolled(double * h_out,
double * h_inLeft,
double * h_inRight,
int numCells,
int numPoints,
bool colMajor) {

	double * d_right;
	double * d_left;
	double * d_out;

	hipMalloc(&d_right, sizeof(double) * numCells  * numPoints);

	hipMalloc(&d_left, sizeof(double) * numCells * numPoints);

	hipMalloc(&d_out, sizeof(double) * numCells);

	hipMemset(d_out, 0, sizeof(double) * numCells);

	hipMemcpy(d_right, h_inRight,
	sizeof(double) * numCells * numPoints, hipMemcpyHostToDevice);

	hipMemcpy(d_left, h_inLeft,
	sizeof(double) * numCells * numPoints, hipMemcpyHostToDevice);


	dim3 blockSize(64);
	dim3 gridSize((numPoints / 64) + 1, (numCells / 64) + 1);

	cudaDocontractDataDataScalar_kernelUnrolled<<<gridSize, blockSize>>>(d_left,
	d_right, d_out, numCells,numPoints);

	hipMemcpy(h_out, d_out, sizeof(double) * numCells, hipMemcpyDeviceToHost);

}


template<class DeviceType, class LeftViewType, class RightViewType, class OutputViewType>
struct contractDataDataScalarFunctor {
	typedef DeviceType device_type;
	LeftViewType _leftFields;
	RightViewType _rightFields;
	OutputViewType _outputFields;
	int _numPoints;

	contractDataDataScalarFunctor(LeftViewType leftFields,
			RightViewType rightFields,
			OutputViewType outputFields,
			int numPoints) :
		_leftFields(leftFields),
		_rightFields(rightFields),
		_outputFields(outputFields),
		_numPoints(numPoints)
	{
		// Nothing to do
	}

	KOKKOS_INLINE_FUNCTION
		void operator()(const unsigned int elementIndex) const {
			double tmpVal = 0;
			for (int qp = 0; qp < _numPoints; qp++) {
				tmpVal += _leftFields(elementIndex, qp)*_rightFields(elementIndex, qp);
			} // D2-loop

			_outputFields(elementIndex) = tmpVal;
		}
};


// Serial contractDataDataScalar.  Contracts FieldContainers of doubles.
void contractDataDataScalarSerial(FieldContainer<double> &  outputFields,
		const FieldContainer<double> &              leftFields,
		const FieldContainer<double> &              rightFields,
		double *                                    time = 0) {

	// TODO(ellen): Might later want to template this so that both the container
	//              and the scalars inside the container are template arguments,
	//              so we can hand it kokkos views or custom structs.
	int numCells      = leftFields.dimension(0);
	int numPoints     = leftFields.dimension(1);

	for (int cl = 0; cl < numCells; cl++) {
		double tmpVal = 0;
		for (int qp = 0; qp < numPoints; qp++) {
			tmpVal += leftFields(cl, qp)*rightFields(cl, qp);
		} // P-loop
		outputFields(cl) = tmpVal;
	} // C-loop
}

/*
 * Kokkos Cuda contractDataDataScalar.
 *
 * Contracts two Kokkos Cuda host views (two double ***** tensors -> one double
 * *** tensor). Since
 *
 * Note that all input and output is in Kokkos host views --- the user is
 * responsible for getting the data in and out of them.
 */
template <class DeviceType, class input_view_t, class output_view_t, class input_host_t, class output_host_t>
void contractDataDataScalarKokkos(output_host_t &   outHost,
		const input_host_t &                      leftHost,
		const input_host_t &                      rightHost,
		output_view_t &                           outDevice,
		input_view_t &                            leftDevice,
		input_view_t &                            rightDevice,
		double *                                  time = 0) {

	// get sizes
	int numCells        = leftHost.dimension(0);
	int numPoints       = leftHost.dimension(1);

	// Deep copy Kokkos host views into device views
	Kokkos::deep_copy(leftDevice, leftHost);
	Kokkos::deep_copy(rightDevice, rightHost);
	Kokkos::deep_copy(outDevice, outHost);

	timespec tic;
	if(time != 0)
		clock_gettime(CLOCK_MONOTONIC, &tic);

	contractDataDataScalarFunctor<DeviceType, input_view_t, input_view_t, output_view_t>
		kokkosFunctor(leftDevice, rightDevice, outDevice, numPoints);

	Kokkos::parallel_for(numCells, kokkosFunctor);

	Kokkos::fence();

	timespec toc;
	if(time !=0){
		clock_gettime(CLOCK_MONOTONIC, &toc);
		*time += getElapsedTime(tic, toc);
	}

	Kokkos::deep_copy(outHost, outDevice);
}

int main(int argc, char* argv[]) {

	int c=1000000, p=24;
  int numRepeats = 10;
	
	FieldContainer<double> inl_c_p(c, p);
	FieldContainer<double> inr_c_p(c, p);
	FieldContainer<double> out1_c(c);
	FieldContainer<double> out2_c(c);
	double zero = Intrepid::INTREPID_TOL*10000.0;
	double temp;
	// fill with random numbers
	for (int i=0; i<inl_c_p.size(); i++) {
		temp = Teuchos::ScalarTraits<double>::random();
		//std::cout << i << " " <<  temp << std::endl;
		inl_c_p[i] = temp;
	}
	for (int i=0; i<inr_c_p.size(); i++) {
		temp = Teuchos::ScalarTraits<double>::random();
		//std::cout << i << " " <<  temp << std::endl;
		inr_c_p[i] = temp;
	}
	std::cout << "Created vectors" << std::endl;

	// ===============================================================
	// ********************** < Kokkos setup> ************************
	// vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

	// Doing all of this here might throw off the timing -- we're not counting the
	// cost of the copy into Kokkos or the deep copy from Kokkos host to Kokkos
	// device.

	Kokkos::initialize();

	// Kokkos Cuda views
	typedef Kokkos::View<double **, Kokkos::LayoutLeft, Kokkos::Cuda> cuda_input_view_t;
	typedef Kokkos::View<double *, Kokkos::LayoutLeft, Kokkos::Cuda> cuda_output_view_t;
	typedef typename cuda_input_view_t::HostMirror cuda_input_host_t;
	typedef typename cuda_output_view_t::HostMirror cuda_output_host_t;

	// Kokkos OpenMP views
	typedef Kokkos::View<double **, Kokkos::LayoutLeft, Kokkos::OpenMP> omp_input_view_t;
	typedef Kokkos::View<double *, Kokkos::LayoutLeft, Kokkos::OpenMP> omp_output_view_t;
	typedef typename omp_input_view_t::HostMirror omp_input_host_t;
	typedef typename omp_output_view_t::HostMirror omp_output_host_t;


	//Cuda arrays
	double * cudaRightColMajor = new double[c * p];
	double * cudaLeftColMajor = new double[c * p];
	double * cudaRightRowMajor = new double[c * p];
	double * cudaLeftRowMajor = new double[c * p];

	double * cudaOut = new double[c];


	// Make equivalent Kokkos views

	cuda_input_view_t cuda_kokkosLeft("left_input", c, p);
	cuda_input_view_t cuda_kokkosRight("right_input", c, p);
	cuda_output_view_t cuda_kokkosOut("output", c);

	omp_input_view_t omp_kokkosLeft("left_input", c, p);
	omp_input_view_t omp_kokkosRight("right_input",  c, p);
	omp_output_view_t omp_kokkosOut("output", c);

	// And their host mirrors

	cuda_input_host_t cuda_hostLeft = Kokkos::create_mirror_view(cuda_kokkosLeft);
	cuda_input_host_t cuda_hostRight = Kokkos::create_mirror_view(cuda_kokkosRight);
	cuda_output_host_t cuda_hostOut = Kokkos::create_mirror_view(cuda_kokkosOut);

	omp_input_host_t omp_hostLeft = Kokkos::create_mirror_view(omp_kokkosLeft);
	omp_input_host_t omp_hostRight = Kokkos::create_mirror_view(omp_kokkosRight);
	omp_output_host_t omp_hostOut = Kokkos::create_mirror_view(omp_kokkosOut);

	// Copy into Kokkos host views and cuda
	// Need to change this so that its 1-D and cl has stride 1
	for (int cl = 0; cl < c; ++cl) {
		for (int qp = 0; qp < p; ++qp) {
			cuda_hostLeft(cl,qp) = inl_c_p(cl,qp);
			omp_hostLeft(cl,qp) = inl_c_p(cl,qp);

			cuda_hostRight(cl,qp) = inr_c_p(cl,qp);
			omp_hostRight(cl,qp) = inr_c_p(cl,qp);

			cudaRightColMajor[cl + c*qp] = inr_c_p(cl,qp);
			cudaLeftColMajor[cl + c*qp] = inl_c_p(cl,qp);

			cudaRightRowMajor[cl * p + qp] = inr_c_p(cl,qp);
			cudaLeftRowMajor[cl * p + qp] = inl_c_p(cl,qp);
		}
	}



	// ===============================================================
	// ********************** </Kokkos setup> ************************
	// vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

	std::cout << "trying serial" << std::endl;

	//Warmup
	contractDataDataScalarSerial(out2_c, inl_c_p, inr_c_p);

	timespec tic;
	clock_gettime(CLOCK_MONOTONIC, &tic);

	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < numRepeats; ++i){
		contractDataDataScalarSerial(out2_c, inl_c_p, inr_c_p);
	}

	timespec toc;
	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_serial = getElapsedTime(tic, toc);

	printf("trying kokkos openmp\n");

	//Warmpup
	contractDataDataScalarKokkos<Kokkos::OpenMP, omp_input_view_t,
		omp_output_view_t, omp_input_host_t, omp_output_host_t>
			(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut,
			 omp_kokkosLeft, omp_kokkosRight);
	clock_gettime(CLOCK_MONOTONIC, &tic);

	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < numRepeats; ++i){
		contractDataDataScalarKokkos<Kokkos::OpenMP, omp_input_view_t,
			omp_output_view_t, omp_input_host_t, omp_output_host_t>
				(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut,
				 omp_kokkosLeft, omp_kokkosRight);
	}

	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_kokkos_omp = getElapsedTime(tic, toc);

	// Copy out from kokkos output view (NOT timing this)
	for (int cl = 0; cl < c; ++cl) {
		//std::cout << omp_hostOut(cl) << std::endl;
		out1_c(cl) = omp_hostOut(cl);
	}

	rst::subtract(&out1_c[0], &out2_c[0], out2_c.size());
	if (rst::vectorNorm(&out1_c[0], out1_c.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check COMP_CPP vs. COMP_KOKKOS; "
			<< " diff-1norm = " << rst::vectorNorm(&out1_c[0], out1_c.size(), Intrepid::NORM_ONE) << "\n\n";
	}

	std::cout << "kokkos omp speedup of " << elapsedTime_serial/elapsedTime_kokkos_omp << std::endl;

	printf("trying kokkos cuda\n");

	//Warmpup
	contractDataDataScalarKokkos<Kokkos::Cuda, cuda_input_view_t,
		cuda_output_view_t, cuda_input_host_t, cuda_output_host_t>
			(cuda_hostOut, cuda_hostLeft, cuda_hostRight, cuda_kokkosOut,
			 cuda_kokkosLeft, cuda_kokkosRight);
	clock_gettime(CLOCK_MONOTONIC, &tic);

	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < numRepeats; ++i){
		contractDataDataScalarKokkos<Kokkos::Cuda, cuda_input_view_t,
			cuda_output_view_t, cuda_input_host_t, cuda_output_host_t>
				(cuda_hostOut, cuda_hostLeft, cuda_hostRight, cuda_kokkosOut,
				 cuda_kokkosLeft, cuda_kokkosRight);
	}

	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_kokkos_cuda = getElapsedTime(tic, toc);

	// Copy out from kokkos output view (NOT timing this)
	for (int cl = 0; cl < c; ++cl) {
		//std::cout << omp_hostOut(cl) << std::endl;
		out1_c(cl) = omp_hostOut(cl);
	}

	rst::subtract(&out1_c[0], &out2_c[0], out2_c.size());
	if (rst::vectorNorm(&out1_c[0], out1_c.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check COMP_CPP vs. COMP_KOKKOS; "
			<< " diff-1norm = " << rst::vectorNorm(&out1_c[0], out1_c.size(), Intrepid::NORM_ONE) << "\n\n";
	}

	std::cout << "kokkos cuda speedup of " << elapsedTime_serial/elapsedTime_kokkos_cuda << std::endl;

	Kokkos::finalize();

	std::cout << "trying cuda col major" << std::endl;
	//Now try the cuda version, start with warmup
	cudaDoContractDataDataScalar(cudaOut,cudaLeftColMajor,cudaRightColMajor, c, p, true);

	clock_gettime(CLOCK_MONOTONIC, &tic);
	for(int i = 0; i < numRepeats; ++i){
		cudaDoContractDataDataScalar(cudaOut,cudaLeftColMajor,cudaRightColMajor, c, p, true);
	}

	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_cuda = getElapsedTime(tic, toc);

	for (int cl = 0; cl < c; ++cl) {
			out1_c(cl) = cudaOut[cl];
	}

	rst::subtract(&out1_c[0], &out2_c[0], out2_c.size());
	if (rst::vectorNorm(&out1_c[0], out1_c.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check cuda; "
		<< " diff-1norm = " << rst::vectorNorm(&out1_c[0], out1_c.size(), Intrepid::NORM_ONE) << "\n\n";
	}

	std::cout << "cuda col major speedup of " << elapsedTime_serial/elapsedTime_cuda << std::endl;

	std::cout << "trying cuda row major" << std::endl;
	//Now try the cuda version, start with warmup
	cudaDoContractDataDataScalar(cudaOut,cudaLeftRowMajor,cudaRightRowMajor, c, p, false);

	clock_gettime(CLOCK_MONOTONIC, &tic);
	for(int i = 0; i < numRepeats; ++i){
		cudaDoContractDataDataScalar(cudaOut,cudaLeftRowMajor,cudaRightRowMajor, c, p, false);
	}

	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_cudaRow = getElapsedTime(tic, toc);

	for (int cl = 0; cl < c; ++cl) {
			out1_c(cl) = cudaOut[cl];
	}

	rst::subtract(&out1_c[0], &out2_c[0], out2_c.size());
	if (rst::vectorNorm(&out1_c[0], out1_c.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check cuda; "
		<< " diff-1norm = " << rst::vectorNorm(&out1_c[0], out1_c.size(), Intrepid::NORM_ONE) << "\n\n";
	}

	std::cout << "cuda row major speedup of " << elapsedTime_serial/elapsedTime_cudaRow << std::endl;

	std::cout << "trying cuda thread-per-index" << std::endl;
	//Now try the cuda version, start with warmup
	cudaDoContractDataDataScalarUnrolled(cudaOut,cudaLeftRowMajor,cudaRightRowMajor, c, p);

	clock_gettime(CLOCK_MONOTONIC, &tic);
	for(int i = 0; i < 5; ++i){
		cudaDoContractDataDataScalarUnrolled(cudaOut,cudaLeftRowMajor,cudaRightRowMajor, c, p);
	}

	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_cudaUnrolled = getElapsedTime(tic, toc);

	for (int cl = 0; cl < c; ++cl) {
		out1_c(cl) = cudaOut[cl];
	}

	rst::subtract(&out1_c[0], &out2_c[0], out2_c.size());
	if (rst::vectorNorm(&out1_c[0], out1_c.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check cuda unrolled; "
		<< " diff-1norm = " << rst::vectorNorm(&out1_c[0], out1_c.size(), Intrepid::NORM_ONE) << "\n\n";
	}

	std::cout << "cuda unrolled speedup of " << elapsedTime_serial/elapsedTime_cudaUnrolled << std::endl;


	return 0;
}
